#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2015, September 7 - October 6
// ###
// ###
// ### Thomas Moellenhoff, Robert Maier, Caner Hazirbas
// ###
// ###
// ###
// ### THIS FILE IS SUPPOSED TO REMAIN UNCHANGED
// ###
// ###


#include "aux.h"
#include <iostream>
#include "math.h"
using namespace std;

// uncomment to use the camera
//#define CAMERA


//CPU Computation


__global__ void mykernel(float *d_imgIn, int h, int nc, size_t pitch, float gamma)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	//d_imgIn[ind_x + ind_y * w + ind_z * w * h] = 0;
	if (x<pitch && y < h && z < nc)
	{
		int index = x + y * pitch + z * pitch * h;
		d_imgIn[index] = powf(d_imgIn[index], gamma);
	}
}



int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;




    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    
    float gamma = 1;
    getParam("gamma", gamma, argc, argv);
    cout << "gamma: " << gamma << endl;

    float blockX = 32;
    getParam("blockX", blockX, argc, argv);
    cout << "blockX: " << blockX << endl;

    float blockY = 8;
    getParam("blockY", blockY, argc, argv);
    cout << "blockY: " << blockY << endl;



    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;




    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    //cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed




    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn  = new float[(size_t)w*h*nc];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[(size_t)w*h*mOut.channels()];




    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);





    Timer timer;
    float t = 1.0f;
    
    float avgTime = 0.0f;
    int imgSize = w*h*nc;
    for (int r=0;r<repeats;r++)
    {
    	timer.start();
    	    // ###
    	    // ###
    	    // ### TODO: Main computation
    	    
    	    int imgSize = w*h*nc;
    	    

    	    	
    	    
    	    for (int i=0;i<imgSize;i++)
    	    {
    			imgOut[i] = pow(imgIn[i],gamma);
    	    }
    	    /*
    	    int nrPixels = w*h;
    	    for (int channel=0; channel<nc; channel++)
    	    {
    	        for (int i=0;i<w;i++)
    	        {
    	        	for (int j=0;j<h;j++)
    	        	{
    	        		imgOut[i+j*w+channel*nrPixels] = pow(imgIn[i+j*w+channel*nrPixels],gamma);
    	          	}
    	        }
    	    }
    	    */
    	    
    	    // ###
    	    // ###
    	    timer.end();  t = timer.get();  // elapsed time in seconds	
    	    avgTime += t;
    	    cout << "time CPU: " << t*1000 << " ms" << endl;
    }
    avgTime = avgTime / repeats;
    cout << "avg time: " << avgTime*1000 << " ms" << endl;


    avgTime = 0.0f;
    for (int r=0;r<repeats;r++)
    {

		//allocate memory on device
		float *d_imgIn;
		size_t pitchInBytes;
        size_t pitch;
		pitchInBytes = hipMallocPitch(&d_imgIn, &pitchInBytes, w, h);
        pitch = pitchInBytes / sizeof(float);
		CUDA_CHECK;
		
		//copy host memory to device
		hipMemcpy2D(d_imgIn, pitch, imgIn, w, w, h, hipMemcpyHostToDevice);
		CUDA_CHECK;
	
		dim3 block = dim3(32,8,1);
		dim3 grid = dim3((w + block.x - 1) / block.x, (h+block.y-1)/block.y, (nc));
        timer.start();

		mykernel <<<grid,block>>> (d_imgIn, h, nc, pitch, gamma);
		timer.end(); t = timer.get();  // elapsed time in seconds

		//copy result back to host memory
		hipMemcpy2D(imgOut, w, d_imgIn, pitch, w, h, hipMemcpyDeviceToHost);
		CUDA_CHECK;
		
		//free memory
		hipFree(d_imgIn);
			
			
		avgTime += t;
		cout << "time GPU: " << t*1000 << " ms" << endl;
    }
    
    avgTime = avgTime / repeats;
    cout << "avg time GPU: " << avgTime*1000 << " ms" << endl;

    //cout << "time: " << t*1000 << " ms" << endl;



    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    convert_layered_to_mat(mOut, imgOut);
    showImage("Output", mOut, 100+w+40, 100);

    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif




    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



