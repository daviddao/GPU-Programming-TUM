#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2015, September 7 - October 6
// ###
// ###
// ### Thomas Moellenhoff, Robert Maier, Caner Hazirbas
// ###
// ###
// ###
// ### THIS FILE IS SUPPOSED TO REMAIN UNCHANGED
// ###
// ###


#include "aux.h"
#include <iostream>
#include "math.h"
using namespace std;

// uncomment to use the camera
//#define CAMERA


//CPU Computation


__global__ void mykernel(float *d_imgIn, int *d_hist, int w, int h, int nc)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    
	if (x < w && y < h && z < nc)
	{
		int ind = x + w*y + w*h*z; 
		int index = d_imgIn[ind]*255.f;
		atomicAdd(&d_hist[index], 1);
	}
}

__global__ void mykernel_shared(float *d_imgIn, int *d_hist, int w, int h, int nc)
{
   __shared__ 
   int shared_hist[256];

   // first thread init the shared memory
   if (threadIdx.x < 256) {
        shared_hist[threadIdx.x] = 0;
   }

   __syncthreads();


   int x = threadIdx.x + blockDim.x * blockIdx.x;
   int y = threadIdx.y + blockDim.y * blockIdx.y;
   int z = threadIdx.z + blockDim.z * blockIdx.z;
   
	if (x < w && y < h && z < nc)
	{
		int ind = x + w*y + w*h*z; 
		int index = d_imgIn[ind]*255.f;
		atomicAdd(&shared_hist[index], 1);
	}

    __syncthreads();

    // first block thread updates global histogramm
    if (threadIdx.x < 256) {
        atomicAdd(&d_hist[threadIdx.x],shared_hist[threadIdx.x]);
    }
}

int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;




    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    
    float gamma = 1;
    getParam("gamma", gamma, argc, argv);
    cout << "gamma: " << gamma << endl;

    float blockX = 32;
    getParam("blockX", blockX, argc, argv);
    cout << "blockX: " << blockX << endl;

    float blockY = 8;
    getParam("blockY", blockY, argc, argv);
    cout << "blockY: " << blockY << endl;



    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;




    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    //cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed




    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn  = new float[(size_t)w*h*nc];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[(size_t)w*h*mOut.channels()];

    int hist_size = 256;
    int *hist = new int[hist_size];
    
	//allocate memory on device
	float *d_imgIn = NULL;
	int *d_hist = NULL;
	int imgSize = (size_t)w*h*nc;
	
	hipMalloc(&d_imgIn, imgSize*sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_hist, hist_size*sizeof(int)); CUDA_CHECK;

    dim3 block = dim3(32, 8, 1);
    dim3 grid = dim3((w + block.x - 1) / block.x,
            (h + block.y - 1) / block.y, (nc));
    
    Timer timer; float t = 0;

    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);
    
    for(int i=0;i<hist_size;i++)
    	hist[i] = 0;
	
	//copy host memory to device
	hipMemcpy(d_imgIn, imgIn, imgSize*sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
	hipMemcpy(d_hist, hist, hist_size*sizeof(int), hipMemcpyHostToDevice); CUDA_CHECK;

    timer.start();
    for (int i=0; i< repeats ; i++)
    {
        
            mykernel_shared <<<grid,block>>> (d_imgIn, d_hist, w, h, nc);
        
    }
    timer.end();
    t = timer.get();  // elapsed time in seconds
    cout << "Average time (shared) for " << repeats << " repeat(s): " << t * 1000 / repeats << " ms" << endl;    

    hipMemcpy(d_hist, hist, hist_size*sizeof(int), hipMemcpyHostToDevice); CUDA_CHECK;
    
    timer.start();
	for (int i=0; i< repeats ; i++)
	{
		
			mykernel <<<grid,block>>> (d_imgIn, d_hist, w, h, nc);
		
	}
    timer.end();
    t = timer.get();  // elapsed time in seconds	
	cout << "Average time (naive) for " << repeats << " repeat(s): " << t * 1000 / repeats << " ms" << endl;  
	
	
//    for(int i=0;i<hist_size;i++)
//    	hist[i] = 0;
//	
//	t = 0;
//	for (int i=0; i< repeats ; i++)
//	{
//		timer.start();
//			mykernel_shared <<<grid,block>>> (d_imgIn, d_hist, w, h, nc);
//		timer.end();
//		t += timer.get();  // elapsed time in seconds
//	}	
//	cout << "SHARED - Average time for " << repeats << " repeat(s): " << t * 1000 / repeats << " ms" << endl;	
	
	//copy result back to host memory
	hipMemcpy(hist, d_hist, hist_size * sizeof(int), hipMemcpyDeviceToHost); CUDA_CHECK;
	
	showHistogram256("Histogram", hist, 1000, 100);
	
    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    //convert_layered_to_mat(mOut, imgOut);
    //showImage("Histogram", mOut, 100+w+40, 100);

    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif

	//free memory
	hipFree(d_imgIn);
	hipFree(d_hist);	


    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;
    delete[] hist;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}




