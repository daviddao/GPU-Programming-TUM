#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2015, September 7 - October 6
// ###
// ###
// ### Thomas Moellenhoff, Robert Maier, Caner Hazirbas
// ###
// ###
// ###
// ### THIS FILE IS SUPPOSED TO REMAIN UNCHANGED
// ###
// ###

#include "aux.h"
#include <iostream>
#include <stdio.h>
#include "math.h"
using namespace std;

// uncomment to use the camera
//#define CAMERA

// calculate eigenvalues
__device__
void calculate2DEigen(float x11, float x12, float x21, float x22, float& eigen1, float& eigen2) {
    
    float tmp1;
    float tmp2;
    // Calculate the trace
    float trace = x11 + x22;
    // Calculate the determinant
    float determinant = x11 * x22 - x12 * x21;

    // Calculate Eigenvalue
    tmp1 = trace / 2.f + sqrt((trace * trace)/(4 - determinant));
    tmp2 = trace / 2.f - sqrt((trace * trace)/(4 - determinant));

    // Sort values
    if (tmp1 > tmp2) {
        eigen1 = tmp2;
        eigen2 = tmp1;
    } else {
        eigen1 = tmp1;
        eigen2 = tmp2;
    }
}

__global__ 
void convolutionkernel(float *d_imgIn, float *d_imgOut, float *d_kernel, int w, int h, int nc, int radius)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x<w && y < h && z < nc)
    {

        int index =x + y * w + z * w * h;
        float res = 0;
        
        d_imgOut[index] = 0;

        for (int diffX = -radius; diffX < radius+1; diffX++) {
            for (int diffY =-radius; diffY < radius+1; diffY++) {
                int tmpi = x + diffX;
                int tmpj = y + diffY;
                if (tmpi<0) tmpi = 0;
                if (tmpi>=w) tmpi = w-1;
                if (tmpj<0) tmpj = 0;
                if (tmpj>=h) tmpj = h-1;
                int indexOffset = tmpi+tmpj*w+z*h*w;
                res += d_imgIn[indexOffset] * d_kernel[(diffX+radius + (2*radius+1)*(diffY+radius))];
            }
        }

        d_imgOut[index] = res;
//      if (x < 10 && y < 10 && z < 2) printf("sum in thread: %d \n", sum);



    }
}

// rotational symmetric derivative discretization for x and y partial derivative
__global__ 
void rotationaldevkernel_x(float *d_imgIn, float *d_imgOut, int w, int h, int nc) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;

    if (x<w && y < h && z < nc)
    {
        int index = x + y * w + z * w * h;
        
        // get the x with clamping
        int x_tmpi = x + 1;
        int x_tmpj = x - 1;

        if (x_tmpi<0) x_tmpi = 0;
        if (x_tmpi>=w) x_tmpi = w-1;

        // get the y with clamping
        int y_tmpi = y + 1;
        int y_tmpj = y - 1;

        if (y_tmpj<0) y_tmpj = 0;
        if (y_tmpj>=h) y_tmpj = h-1;

        // assign the discretisation result to the pixel
        d_imgOut[index] =   3*d_imgIn[x_tmpi + y_tmpi * w + z * w * h]
                            + 10*d_imgIn[x_tmpi + y * w + z * w * h]
                            + 3*d_imgIn[x_tmpi + y_tmpj * w + z * w * h]
                            - 3*d_imgIn[x_tmpj + y_tmpi * w + z * w * h]
                            - 10*d_imgIn[x_tmpj + y * w + z * w * h]
                            - 3*d_imgIn[x_tmpj + y_tmpj * w + z * w * h];

        d_imgOut[index] /= 32.f;
    }
}

__global__ 
void rotationaldevkernel_y(float *d_imgIn, float *d_imgOut, int w, int h, int nc) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;

    if (x<w && y < h && z < nc)
    {
        int index = x + y * w + z * w * h;
        
        // get the x with clamping
        int x_tmpi = x + 1;
        int x_tmpj = x - 1;

        if (x_tmpi<0) x_tmpi = 0;
        if (x_tmpi>=w) x_tmpi = w-1;

        // get the y with clamping
        int y_tmpi = y + 1;
        int y_tmpj = y - 1;

        if (y_tmpj<0) y_tmpj = 0;
        if (y_tmpj>=h) y_tmpj = h-1;

        // assign the discretisation result to the pixel
        d_imgOut[index] =   3*d_imgIn[x_tmpi + y_tmpi * w + z * w * h]
                            + 10*d_imgIn[x + y_tmpi * w + z * w * h]
                            + 3*d_imgIn[x_tmpj + y_tmpi * w + z * w * h]
                            - 3*d_imgIn[x_tmpi + y_tmpj * w + z * w * h]
                            - 10*d_imgIn[x + y_tmpj * w + z * w * h]
                            - 3*d_imgIn[x_tmpj + y_tmpj * w + z * w * h];

        d_imgOut[index] /= 32.f;
    }
}

__global__
void calculate_M(float *d_xOut, float *d_yOut, float* d_m1, float* d_m2, float* d_m3, int w, int h, int nc) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;

    if (x<w && y < h && z < nc)
    {   
        float res_m1 = 0;
        float res_m2 = 0;
        float res_m3 = 0;
        int index;
        for (int i = 0; i < nc; i++) {
            int index = x + y * w + i * w * h;
            res_m1 += (d_xOut[index] * d_xOut[index]);
            res_m2 += (d_yOut[index] * d_yOut[index]);
            res_m3 += (d_yOut[index] * d_xOut[index]);

        }

        index = x + y * w + z * w * h;
        d_m1[index] = res_m1;
        d_m2[index] = res_m2;
        d_m3[index] = res_m3;
    }
}

__global__
void edgedetector(float* m1, float* m2, float* m3, float* edgeOut, int w, int h, int nc, float alpha, float beta) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;

    float eigen1;
    float eigen2;

    if (x<w && y < h && z < nc)
    {   
        int index = x + y * w + z * w * h;
        float x11 = m1[index];
        float x22 = m2[index];
        float x12 = m3[index];
        float x21 = x12;


        int redIndex = x + y * w;
        int greenIndex = x + y * w + 1 * w * h;
        int blueIndex = x + y * w + 2 * w * h;

        // calculate the eigenvalues
        calculate2DEigen(x11, x12, x21, x22, eigen1, eigen2);

        if (eigen1 >= alpha) {
            //Corner, make it red!

            edgeOut[redIndex] = 255;
            edgeOut[greenIndex] = 0;
            edgeOut[blueIndex] = 0;

        } else if (eigen1 <= beta && alpha <= eigen2) {
            //Edge, draw green!

            edgeOut[redIndex] = 0;
            edgeOut[greenIndex] = 255;
            edgeOut[blueIndex] = 0;

        } else {
            edgeOut[index] *= 0.5;
        }
    }
}


int main(int argc, char **argv) {
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();
    CUDA_CHECK;

    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret)
        cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) {
        cout << "Usage: " << argv[0]
                << " -i <image> [-repeats <repeats>] [-gray]" << endl;
        return 1;
    }
#endif

    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;

    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    float sigma = 2;
    getParam("sigma", sigma, argc, argv);
    cout << "sigma: " << sigma << endl;

    float scale = 300;
    getParam("scale", scale, argc, argv);
    cout << "scale: " << scale << endl;

    float alpha = 0.01;
    getParam("alpha", alpha, argc, argv);
    cout << "alpha: " << alpha << endl;

    float beta = 0.001;
    getParam("beta", beta, argc, argv);
    cout << "beta: " << beta << endl;
    // ### Define your own parameters here as needed    

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
    cv::VideoCapture camera(0);
    if(!camera.isOpened()) {cerr << "ERROR: Could not open camera" << endl; return 1;}
    int camW = 640;
    int camH = 480;
    camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
    camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;

#else

    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(),
            (gray ? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) {
        cerr << "ERROR: Could not load image " << image << endl;
        return 1;
    }

#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn, CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;

    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h, w, mIn.type()); // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    //cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed
    cv::Mat m11(h, w, mIn.type());
    cv::Mat m12(h, w, mIn.type());
    cv::Mat m13(h, w, mIn.type());

    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn = new float[(size_t) w * h * nc];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[(size_t) w * h * mOut.channels()];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *m1Out = new float[(size_t) w * h * mOut.channels()];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *m2Out = new float[(size_t) w * h * mOut.channels()];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *m3Out = new float[(size_t) w * h * mOut.channels()];

    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
        // Get camera image
        camera >> mIn;
        // convert to float representation (opencv loads image values as single bytes by default)
        mIn.convertTo(mIn,CV_32F);
        // convert range of each channel to [0,1] (opencv default is [0,255])
        mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered(imgIn, mIn);

    Timer timer;
    timer.start();
    // ###
    // ###
    // ### TODO: Main computation

    //compute gaussian kernel
//  float sigma = 2;
    int radius = ceil(3 * sigma);
    float factor = 1 / (2 * M_PI * sigma * sigma);
    //float gaussian[(2*radius+1)*(2*radius+1)];
    int kernelsize = (2*radius+1)*(2*radius+1);
    float* gaussian = new float[kernelsize];
    int diffX;
    int diffY;
    float entry;
    float sum = 0;
    for (int i = 0; i < 2 * radius+1; i++) {
        for (int j=0; j<2*radius+1; j++) {
            diffX=i-radius;
            diffY=j-radius;
            entry = factor * exp(-((diffX*diffX+diffY*diffY)/(2*sigma*sigma)));
            if (entry < 0.001f) entry = 0;
            gaussian[i+(2*radius+1)*j]=entry;
            sum += entry;
        }
    }
    //normalize kernel
    for (int j = 0; j < 2 * radius+1; j++) {
        for (int i=0; i< 2 *radius+1; i++) {
            gaussian[i + (2*radius+1)*j] = gaussian[i + (2*radius+1)*j]/sum;
            cout <<  gaussian[i + (2*radius+1)*j] << "  ";
        }
        cout << endl;
    }


    //create kernel
    cv::Mat kernel(radius*2+1, radius*2+1, CV_32FC1, gaussian);
    
    //make max value 1
//  for (int j = 0; j < 2 * radius+1; j++) {
//      for (int i=0; i< 2 *radius+1; i++) {
//          gaussian[i + (2*radius+1)*j] = gaussian[i + (2*radius+1)*j]/gaussian[radius + (2*radius+1)*radius];
//      }
//  }
//  cv::Mat kernel(radius*2+1, radius*2+1, CV_32FC1, gaussian);

    
    //normalize kernel
    //cv::normalize(kernel, kernel, 1);
    showImage("Kernel", kernel, 0,0);
    
    
//  sum = 0;
//  for (int j = 0; j < 2 * radius+1; j++) {
//      for (int i=0; i< 2 *radius+1; i++) {
//          sum += gaussian[i + (2*radius+1)*j];
//      }
//  }
//  cout << "sum: " << sum << endl;
    
    
    for (int i = 0; i < w*h*nc; i++){
        imgOut[i]=0;
    }

/*
    //implement convolution
    for (int c=0; c<nc; c++){
        for (int i=0; i<w; i++){
            for (int j=0; j<h; j++){
                int index = i+j*w+c*h*w;
                
                for (int diffX = -radius; diffX < radius+1; diffX++) {
                    for (int diffY =-radius; diffY < radius+1; diffY++) {
                        int tmpi = i + diffX;
                        int tmpj = j + diffY;
                        if (tmpi<0) tmpi = 0;
                        if (tmpi>=w) tmpi = w-1;
                        if (tmpj<0) tmpj = 0;
                        if (tmpj>=h) tmpj = h-1;
                        int indexOffset = tmpi+tmpj*w+c*h*w;
                        
                        imgOut[index] += imgIn[indexOffset] * gaussian[(diffX+radius + (2*radius+1)*(diffY+radius))];
                    }
                }
                
            }
        }
    }
*/
    //Convolute on GPU
    //allocate memory on device
    float *d_imgIn;
    float *d_imgOut;
    float *d_kernel;
    float *d_xOut;
    float *d_yOut;
    float *d_m1;
    float *d_m2;
    float *d_m3;
    float *d_m1Out;
    float *d_m2Out;
    float *d_m3Out;
    int imgSize = w*h*nc;
    hipMalloc(&d_imgIn, imgSize * sizeof(float));
    hipMalloc(&d_imgOut, imgSize * sizeof(float));
    hipMalloc(&d_kernel, kernelsize * sizeof(float));
    // Storing the partial derivatives
    hipMalloc(&d_xOut, imgSize * sizeof(float));
    hipMalloc(&d_yOut, imgSize * sizeof(float));
    // Storing m11, m12, m13
    hipMalloc(&d_m1, imgSize * sizeof(float));
    hipMalloc(&d_m2, imgSize * sizeof(float));
    hipMalloc(&d_m3, imgSize * sizeof(float));
    // Storing the convolved m11, m12, m13
    hipMalloc(&d_m1Out, imgSize * sizeof(float));
    hipMalloc(&d_m2Out, imgSize * sizeof(float));
    hipMalloc(&d_m3Out, imgSize * sizeof(float));
    CUDA_CHECK;

//  hipDeviceSynchronize();
    hipMemcpy(d_imgIn, imgIn, imgSize * sizeof(float),
            hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, gaussian, kernelsize * sizeof(float),
            hipMemcpyHostToDevice);
    
    CUDA_CHECK;

    dim3 block = dim3(32, 8, 1);
    dim3 grid = dim3((w + block.x - 1) / block.x,
            (h + block.y - 1) / block.y, (nc));

    convolutionkernel <<<grid,block>>> (d_imgIn, d_imgOut, d_kernel, w, h, nc, radius);

    // calculate the partial derivatives of the blurred image
    rotationaldevkernel_x <<<grid,block>>> (d_imgOut, d_xOut , w, h, nc);
    rotationaldevkernel_y <<<grid,block>>> (d_imgOut, d_yOut, w, h, nc);

    // now calculate M
    calculate_M <<<grid,block>>> (d_xOut, d_yOut, d_m1, d_m2, d_m3, w, h, nc);

    convolutionkernel <<<grid,block>>> (d_m1, d_m1Out, d_kernel, w, h, nc, radius);
    convolutionkernel <<<grid,block>>> (d_m2, d_m2Out, d_kernel, w, h, nc, radius);
    convolutionkernel <<<grid,block>>> (d_m3, d_m3Out, d_kernel, w, h, nc, radius);

    // copy the input image
    hipMemcpy(d_imgOut, imgIn, imgSize * sizeof(float),
            hipMemcpyHostToDevice);

    // edge detector
    edgedetector <<<grid,block>>> (d_m1Out, d_m2Out, d_m3Out, d_imgOut, w, h, nc, alpha, beta);

    hipMemcpy(imgOut, d_imgOut, imgSize * sizeof(float),
            hipMemcpyDeviceToHost);
    hipMemcpy(m1Out, d_m1Out, imgSize * sizeof(float),
            hipMemcpyDeviceToHost);
    hipMemcpy(m2Out, d_m2Out, imgSize * sizeof(float),
            hipMemcpyDeviceToHost);
    hipMemcpy(m3Out, d_m3Out, imgSize * sizeof(float),
            hipMemcpyDeviceToHost);
    CUDA_CHECK;
    
    //free memory
    hipFree(d_imgIn);
    hipFree(d_imgOut);
    hipFree(d_kernel);
    
    
// ###
// ###
timer.end();
float t = timer.get();  // elapsed time in seconds
cout << "time: " << t * 1000 << " ms" << endl;

  // show input image
showImage("Input", mIn, 100, 100); // show at position (x_from_left=100,y_from_above=100)

 // show output image: first convert to interleaved opencv format from the layered raw array
convert_layered_to_mat(mOut, imgOut);
showImage("Output", mOut, 100 + w + 40, 100);


// ### Display your own output images here as needed
convert_layered_to_mat(m11, m1Out);
m11 *= scale;
showImage("m1", m11, 100 + w + 60, 100);

convert_layered_to_mat(m12, m2Out);
m12 *= scale;
showImage("m2", m12, 100 + w + 80, 100);

convert_layered_to_mat(m13, m3Out);
m13 *= scale;
showImage("m3", m13, 100 + w + 100, 100);



#ifdef CAMERA
 // end of camera loop
}
#else
 // wait for key inputs
cv::waitKey(0);
#endif

 // save input and result
cv::imwrite("image_input.png", mIn * 255.f); // "imwrite" assumes channel range [0,255]
cv::imwrite("image_result.png", mOut * 255.f);

 // free allocated arrays
delete[] imgIn;
delete[] imgOut;
delete[] gaussian;

 // close all opencv windows
cvDestroyAllWindows();
return 0;
}
