#include "hip/hip_runtime.h"
/**
 * David Dao, Johannes Rausch, Michal Szymczak
 * TU Munich
 * Sep 2015
 */

#include <stdio.h>
#include "mex.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipblas.h>
using namespace std;

#define	max(A, B)	((A) > (B) ? (A) : (B))
#define	min(A, B)	((A) < (B) ? (A) : (B))

// cuda error checking
string prev_file = "";
int prev_line = 0;
#define CUDA_CHECK cuda_check(__FILE__,__LINE__)
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        if (prev_line>0) cout << "Previous CUDA call:" << endl << prev_file << ", line " << prev_line << endl;
        exit(1);
    }
    prev_file = file;
    prev_line = line;
}


/* Input arguments */
#define IN_A		prhs[0]
#define IN_B		prhs[1]

/* Output arguments */
#define OUT_X		plhs[0]

/* Gateway routine */
void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[] )
{
  double *matrix_A, *matrix_B, *matrix_X;
  int     N, D;
  
  /* Get the sizes of each input argument */
  N = mxGetM(IN_A);
  D = mxGetN(IN_B);
  
  /* Create the new arrays and set the output pointers to them */
  //OUT_X     = mxCreateDoubleMatrix(N, D, mxREAL);
    OUT_X     = mxCreateDoubleMatrix(N, N, mxREAL);


    /* Assign pointers to the input arguments */
  matrix_A      = mxGetPr(IN_A);
  matrix_B      = mxGetPr(IN_B);
  
  /* Assign pointers to the output arguments */
  matrix_X      = mxGetPr(OUT_X);
  
  /* Do the actual computations in a subroutine */
    //allocate memory on GPU
    double* d_matrix_A;
    double* d_matrix_X;
    double *d_Aarray = new double[1];
    double **dd_Aarray;
    dd_Aarray = (double**)malloc(sizeof(double*));
    int* d_infoArray;
    
    
    hipblasStatus_t status;
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        printf("error1 %i\n",status);
        return;
    }

    hipMalloc((void**)&dd_Aarray[0], N*N*sizeof(double)); CUDA_CHECK;
    hipMemcpy(dd_Aarray[0], matrix_A, N * N * sizeof(double), hipMemcpyHostToDevice); CUDA_CHECK;
    
    //hipMalloc(&d_matrix_A, N * N * sizeof(double)); CUDA_CHECK;
   // d_Aarray = d_matrix_A;

    hipMalloc(&d_matrix_X, N * N * sizeof(double)); CUDA_CHECK;
    hipMalloc(&d_infoArray, N * N * sizeof(int)); CUDA_CHECK;   
   

    //hipMemcpy(d_matrix_A, matrix_A, N * N * sizeof(double), hipMemcpyHostToDevice); CUDA_CHECK;
    //hipMemcpy(dd_Aarray, d_Aarray, sizeof(double*), hipMemcpyDeviceToDevice); CUDA_CHECK;

   /* for(int i=0;i<N;i++)
    {
        for( int j=0;j<N;j++)
        {
            printf("%.2f ", matrix_A[i*N + j]);
        }
    printf("\n");
    }*/

    status = hipblasDgetrfBatched(handle,
                               N, 
                               dd_Aarray,
                               N, 
                               NULL,
                               d_infoArray,
                               1);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        printf("error2 %i\n",status);
        return;
    }

    hipMemcpy(matrix_X, dd_Aarray[0], N * sizeof(double), hipMemcpyDeviceToHost);  CUDA_CHECK;

    hipblasDestroy(handle);
    hipFree(d_matrix_A);
    hipFree(d_matrix_X);
    delete[] d_Aarray;
    free(dd_Aarray);
    
  return;
}



