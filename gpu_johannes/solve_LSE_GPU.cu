#include "hip/hip_runtime.h"
/**
 * David Dao, Johannes Rausch, Michal Szymczak
 * TU Munich
 * Sep 2015
 */

#include <stdio.h>
#include "mex.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipblas.h>
using namespace std;

#define	max(A, B)	((A) > (B) ? (A) : (B))
#define	min(A, B)	((A) < (B) ? (A) : (B))

// cuda error checking
string prev_file = "";
int prev_line = 0;
#define CUDA_CHECK cuda_check(__FILE__,__LINE__)
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        if (prev_line>0) cout << "Previous CUDA call:" << endl << prev_file << ", line " << prev_line << endl;
        //system("PAUSE");
    }
    prev_file = file;
    prev_line = line;
}








/* Input arguments */
#define IN_A		prhs[0]
#define IN_B		prhs[1]

/* Output arguments */
#define OUT_X		plhs[0]

/* Gateway routine */
void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[] )
{
  double *matrix_A, *matrix_B, *matrix_X;
  int     N, D;
  
  /* Get the sizes of each input argument */
  N = mxGetM(IN_A);
  D = mxGetN(IN_B);
  
  /* Create the new arrays and set the output pointers to them */
  //OUT_X     = mxCreateDoubleMatrix(N, D, mxREAL);
  OUT_X     = mxCreateDoubleMatrix(N, N, mxREAL);


    /* Assign pointers to the input arguments */
  matrix_A      = mxGetPr(IN_A);
  matrix_B      = mxGetPr(IN_B);
  
  /* Assign pointers to the output arguments */
  matrix_X      = mxGetPr(OUT_X);
  
  /* Do the actual computations in a subroutine */
    //allocate memory on GPU
    double** Aarray = (double**)malloc(sizeof(double*));;
    int* d_infoArray;
    int *infoArray = (int*)malloc(sizeof(int));
    
    hipblasStatus_t status;
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        printf("error for create the handle %i\n",status);
        goto ENDING;
        //return;
    }

    hipMalloc((void**)&Aarray[0], N * N * sizeof(double)); CUDA_CHECK;
    hipMemcpy(Aarray[0], matrix_A, N * N * sizeof(double), hipMemcpyHostToDevice); CUDA_CHECK;
    
    double** d_Aarray;
    hipMalloc(&d_Aarray, sizeof(double*));
    hipMemcpy(d_Aarray, Aarray, sizeof(double*), hipMemcpyHostToDevice); CUDA_CHECK;

    //hipMalloc(&d_matrix_X, N * N * sizeof(double)); CUDA_CHECK;
    hipMalloc(&d_infoArray, N * N * sizeof(int)); CUDA_CHECK;   



    status = hipblasDgetrfBatched(handle, N, d_Aarray, N, NULL, d_infoArray, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        printf("error for Batched fun %i\n",status);
        goto ENDING;
        //return;
    }

    hipMemcpy(matrix_X, Aarray[0], N * N * sizeof(double), hipMemcpyDeviceToHost);  CUDA_CHECK;
    hipMemcpy(infoArray, d_infoArray, sizeof(int), hipMemcpyDeviceToHost);  CUDA_CHECK;    
    
    printf("LU decomposition status (0 - successsful): %d", infoArray[0]);

ENDING:
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        printf("error for destroy the handle %i\n",status);
        //return;
    }
    free(Aarray);
    free(infoArray);
  return;
}



