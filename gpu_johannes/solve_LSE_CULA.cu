/**
 * David Dao, Johannes Rausch, Michal Szymczak
 * TU Munich
 * Sep 2015
 */

#include <stdio.h>
#include "mex.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cula_device.h>
#include <cula_lapack_device.h>

using namespace std;

void checkStatus(culaStatus status)
{
char buf[80];

if(!status)
return;

culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
printf("%s\n %d", buf, status);

culaShutdown();
//exit(EXIT_FAILURE);
}

// cuda error checking
string prev_file = "";
int prev_line = 0;
#define CUDA_CHECK cuda_check(__FILE__,__LINE__)
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        if (prev_line>0) cout << "Previous CUDA call:" << endl << prev_file << ", line " << prev_line << endl;
        hipDeviceReset();
        //exit(0);
    }
    prev_file = file;
    prev_line = line;
}

/* Input arguments */
#define IN_A		prhs[0]
#define IN_B		prhs[1]
//#define IN_X		prhs[2]

/* Output arguments */
#define OUT_X		plhs[0]

/* Gateway routine */
void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[] )
{
  double *matrix_A, *matrix_B, *matrix_X;
  double *d_matrix_A, *d_matrix_B, *d_matrix_X;
  int     N, D;
  int iter = 0;
  
  /* Get the sizes of each input argument */
  N = mxGetM(IN_A);
  D = mxGetN(IN_B);
  
    /* Assign pointers to the input arguments */
  matrix_A      = mxGetPr(IN_A);
  matrix_B      = mxGetPr(IN_B);
 // matrix_X      = mxGetPr(IN_X);
  
  /* Create the new arrays and set the output pointers to them */
  //OUT_X     = mxCreateDoubleMatrix(N, D, mxREAL);
  
  /* Assign pointers to the output arguments */
  //W      = mxGetPr(OUT_X);

  int matrix_pivot[N];
  int *d_matrix_pivot;
  culaStatus status;
  status = culaInitialize();
  checkStatus(status);
  
  hipMalloc(&d_matrix_A, N * N * sizeof(double)); CUDA_CHECK;
  hipMalloc(&d_matrix_B, N * D * sizeof(double)); CUDA_CHECK;
 // hipMalloc(&d_matrix_X, N * D * sizeof(double)); CUDA_CHECK;  
  hipMalloc(&d_matrix_pivot, N * sizeof(int)); CUDA_CHECK;
  hipMemcpy(d_matrix_A, matrix_A, N * N * sizeof(double), hipMemcpyHostToDevice); CUDA_CHECK;
  hipMemcpy(d_matrix_B, matrix_B, N * D * sizeof(double), hipMemcpyHostToDevice); CUDA_CHECK;
 // hipMemcpy(d_matrix_X, matrix_X, N * D * sizeof(double), hipMemcpyHostToDevice); CUDA_CHECK;
  
  //status = culaDgetrs(char trans, int n, int nrhs, culaDouble* a, int lda, culaInt* ipiv, culaDouble* b, int ldb);
  printf("N: %d, D: %d\n", N, D);
  
  //status = culaDeviceDgetrf(N, N, d_matrix_A, N, d_matrix_pivot);
//  status = culaDeviceDgetrf(N, N, d_matrix_A, N, d_matrix_pivot);
//  checkStatus(status);
 // status = culaDeviceDsgesv(N, D, d_matrix_A, N, d_matrix_pivot, d_matrix_B, N, d_matrix_X, N, &iter);
    status = culaDeviceDgesv(N, D, d_matrix_A, N, d_matrix_pivot, d_matrix_B, N);
 // status = culaDeviceDgetrs('N', N, D, d_matrix_A, N, d_matrix_pivot, d_matrix_B, N);

//  status = culaDeviceDgetrs('N', N, D, d_matrix_A, N, d_matrix_pivot, d_matrix_B, N);
  checkStatus(status);
  culaShutdown();
  
  hipMemcpy(matrix_X, d_matrix_X, N * D * sizeof(double), hipMemcpyDeviceToHost); CUDA_CHECK;
  
  hipFree(d_matrix_A);
  hipFree(d_matrix_B);
//  hipFree(d_matrix_X);
  hipFree(d_matrix_pivot);
  
  return;
}



