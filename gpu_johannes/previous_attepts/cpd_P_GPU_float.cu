#include "hip/hip_runtime.h"
/**magicCUDA('cpd_P_GPU')magicCUDA('cpd_P_GPU')
 * Johannes and David
 * TU Munich
 * Sep 2015
 */
#include <stdio.h>
#include "mex.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <ctime>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "time.h"
#include <cstdlib>
#include <iostream>
//using std::string;
//using std::cout;
//using std::endl;

#define IDX2C(i,j,ld) (((j)*(ld))+(i)) //modify index for 0-based indexing

#define	max(A, B)	((A) > (B) ? (A) : (B))
#define	min(A, B)	((A) < (B) ? (A) : (B))


// error check macros
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

// for CUBLAS V2 API
#define cublasCheckErrors(fn) \
    do { \
        hipblasStatus_t __err = fn; \
        if (__err != HIPBLAS_STATUS_SUCCESS) { \
            fprintf(stderr, "Fatal cublas error: %d (at %s:%d)\n", \
                (int)(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

// data filler
void fillvector(float *data, int N, float value){
    for(int i=0; i<N; i++){
        data[i] = value;
    }
}

// measuring time 
class Timer
{
    public:
	Timer() : tStart(0), running(false), sec(0.f)
	{
	}
	void start()
	{
		tStart = clock();
		running = true;
	}
	void end()
	{
		if (!running) { sec = 0; return; }
        hipDeviceSynchronize();
		clock_t tEnd = clock();
		sec = (float)(tEnd - tStart) / CLOCKS_PER_SEC;
		running = false;
	}
	float get()
	{
		if (running) end();
		return sec;
	}
    private:
	clock_t tStart;
	bool running;
	float sec;
};

// Calculates Pt1 using threads
__global__
void calc_Pt1(float* d_Pt1, float* d_sp, float outlier_tmp, int N) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    
    if (idx < N) {
        d_Pt1[idx] = 1.0f - (outlier_tmp/(d_sp[idx] + outlier_tmp));
    }
}
// Use threads to calculate E, later we sum up
__global__
void calc_E(float* d_E, float* d_sp, float outlier_tmp, int N) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    
    if (idx < N) {
        d_E[idx] = -log(d_sp[idx] + outlier_tmp);
    }
}

// Calculates Px using threads
__global__
void calc_X_tmp(float* d_Xtemp, float* d_X, float* d_denom, int starting_index, int slice_size, int D, int N) {
    
     int idx = threadIdx.x + blockDim.x * blockIdx.x;
     int d = threadIdx.y + blockDim.y * blockIdx.y;
    
     if (idx < slice_size && d < D) {
        // Create d_Xtemp (slice_size * D)
        d_Xtemp[IDX2C(idx,d,slice_size)] = d_denom[idx] * d_X[IDX2C(idx + starting_index,d,N)];   
     } 
}
    
// Calculates slice_size denominator using threads
__global__
void calc_denominator(float* d_denom, float* d_sp, float outlier_tmp, int slice_size) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    
    if (idx < slice_size) {
        d_denom[idx] = 1.0f / (d_sp[idx] + outlier_tmp);
    }
}

// Kernel calculating the nominators of each entry of P (for 6980 x 6980 it takes 160ms)
__global__ 
void calc_nominator(float* d_X, float* d_Y, float* d_PSlice, float ksig, int N, int M, int D, int slice_size, int slice_nr){
	
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
    
	int i = idx + (slice_size*slice_nr);
	if (idx < slice_size && i<N && j < M){
		
		
		float diff = 0;
		float razn = 0;
		for (int d=0; d < D; d++) { //iterate through D dimensions
        	
            diff=d_X[i+d*N] - d_Y[j+d*M];
            diff=diff*diff; //take the square of the euclidean norm of one scalar -> square the scalar
            razn+=diff; //proposed name: eucl_dist_sqr; add up the differences for each dimension to get the scalar length of the high-dimensional vector
        }
        // Set it using a row-major -> column major translator (for CUBLAS and MATLAB)
		d_PSlice[IDX2C(i%slice_size,j,slice_size)]=exp(razn/ksig); //nominator
		
	}
}


void cpd_comp(
		float* x,
		float* y, 
        float* sigma2,
		float* outlier,
        float* P1,
        float* Pt1,
        float* Px,
    	float* E,
        int N,
		int M,
        int D
        )
{
  int		n, m, d;
  float	ksig, diff, razn, outlier_tmp, sp;
  float	*P, *temp_x;
  float *PSlice;
  int slice_size = N/10;
  float *ones;
  float *filler;
  
  P = (float*) calloc(M, sizeof(float));
  temp_x = (float*) calloc(D, sizeof(float));
  PSlice = (float*) calloc(slice_size*M, sizeof(float));
  ones = (float*) calloc(M, sizeof(float));
  filler = (float*) calloc(N,sizeof(float));
  ksig = -2.0 * *sigma2;
  //outlier_tmp=(*outlier*M*pow (-ksig*M_PI,0.5*D))/((1-*outlier)*N);
  outlier_tmp=(*outlier*pow (-ksig*M_PI,0.5*D))/((1-*outlier));
  
  // Trick like for Andrey
  if(outlier_tmp == 0) outlier_tmp = 1e-38;
  //mexPrintf("OUTLIER %.64f \n", outlier_tmp);
  
  fillvector(ones, M, 1);
  fillvector(filler,N,0);
  
 /* printf ("ksig = %lf\n", *sigma2);*/
  /* outlier_tmp=*outlier*N/(1- *outlier)/M*(-ksig*3.14159265358979); */
  
  
  
  // CUBLAS Stuff
  hipError_t cudaStat;    
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  
  float* d_X;
  float* d_Y;
  float* d_PSlice; 
  float* d_PSlice_mat; 
  float* d_P1;
  float* d_P1_tmp;
  float* d_Pt1;
  float* d_Px;
  float* d_E;
  float* d_ones;
  float* slice_tmp;
  slice_tmp = (float *)malloc(M*D*sizeof(float));
  float* d_sp;
  
  float* d_denom; //stores a denominator vector
  float* d_X_tmp; //stores a sliced X * denom version of X
  
  
  //TODO: Finish Matrix Vector Multiplication
 
  // Allocate memory on the device
  hipMalloc (&d_X, N*D*sizeof(float));
  hipMalloc (&d_Y, M*D*sizeof(float));
  hipMalloc (&d_PSlice, M*slice_size*sizeof(float));
  
  hipMalloc (&d_P1, N*sizeof(float));
  hipMalloc (&d_P1_tmp, N*sizeof(float));
  
  hipMalloc (&d_Pt1, M*sizeof(float));
  hipMalloc (&d_Px, M*D*sizeof(float));
  hipMalloc (&d_E, N*sizeof(float));
  hipMalloc (&d_ones, M * sizeof(float));
  hipMalloc (&d_sp, N*sizeof(float));
  
  hipMalloc (&d_denom, slice_size*sizeof(float));
  hipMalloc (&d_X_tmp, slice_size*D*sizeof(float));

  cudaCheckErrors("cuda malloc fail");
  
  // Create CUBLAS Context
  stat = hipblasCreate(&handle);
  
  // TODO: Load data in the beginning instead of every time!
  hipMemcpy(d_X,  x, N*D* sizeof(float), hipMemcpyHostToDevice);  
  hipMemcpy(d_Y,  y, M*D* sizeof(float), hipMemcpyHostToDevice);  
  hipMemcpy(d_ones,  ones, M*sizeof(float), hipMemcpyHostToDevice);  
  hipMemcpy(d_sp,  filler, N*sizeof(float), hipMemcpyHostToDevice);
  // Cpy Px to GPU once!
  hipMemcpy(d_Px, Px, N*D*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_P1, P1, N*sizeof(float),hipMemcpyHostToDevice);

  int numSlices = N / slice_size;
  
  dim3 block;
  dim3 grid;
  
  block = dim3(4, 32, 1);
  grid = dim3((slice_size + block.x - 1) / block.x,
			(M + block.y - 1) / block.y);

  
  
  Timer timer; timer.start();
  for (int s=0; s<numSlices; s++){
	  
      //mexPrintf("\n Iteration %i \n",s);
      
      block = dim3(4, 32, 1);
      grid = dim3((slice_size + block.x - 1) / block.x,
			(M + block.y - 1) / block.y);
      
	  calc_nominator <<<grid, block>>> (d_X, d_Y, d_PSlice, ksig, N, M, D, slice_size, s); 
	   
	   float alpha = 1.0f;
	   float beta = 0.0f;
	   int rowsA = slice_size;
	   int columnsA = M;
	   
       // Calculates sp without outlier
       stat = hipblasSgemv(handle, HIPBLAS_OP_N, rowsA, columnsA, &alpha, d_PSlice, slice_size, d_ones, 1, &beta, d_sp+(s*slice_size), 1);
	   cublasCheckErrors(stat);
       
       // Get the denominator as 1/sp + outlier in d_denom
       block = dim3(256, 1, 1);
       grid = dim3((slice_size + block.x - 1) / block.x,1);
       // denominator correctly calculates! (tested for 6890)
       calc_denominator  <<<grid, block>>> (d_denom, d_sp+(s*slice_size), outlier_tmp, slice_size);
       
       // Calculate P1 using PSlice_t * denom
       stat = hipblasSgemv(handle, HIPBLAS_OP_T, rowsA, columnsA, &alpha, d_PSlice, slice_size,  d_denom, 1, &beta, d_P1_tmp, 1);
       cublasCheckErrors(stat);
       
       // Add P1_tmp to P1
       stat = hipblasSaxpy(handle, M, &alpha, d_P1_tmp, 1, d_P1, 1);
       cublasCheckErrors(stat);
       
       // Calculate Px
       block = dim3(64, 4, 1);
       grid = dim3((slice_size + block.x - 1) / block.x,
			(D + block.y - 1) / block.y);
       
       // First calculate X_temp_sliced (takes 50ms)
       calc_X_tmp <<<grid, block>>> (d_X_tmp, d_X, d_denom, (s*slice_size), slice_size, D, N); 
       
       // Do PSlice_t * X_tmp =+ Px 
       beta = 1.0f;
       stat = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, D, slice_size, &alpha, d_PSlice, slice_size, d_X_tmp, slice_size, &beta, d_Px, M);
       cublasCheckErrors(stat);
	   
  }
  
  // Calculates the complete P1
  block = dim3(256, 1, 1);
  grid = dim3((N + block.x - 1) / block.x,1);
  calc_Pt1  <<<grid, block>>> (d_Pt1, d_sp, outlier_tmp, N);
  // Calculate E
  calc_E  <<<grid, block>>> (d_E, d_sp, outlier_tmp, N);
  // Sum up E
  stat = hipblasSasum(handle, N, d_E, 1, &*E);
    
  *E +=D*N*-log(*sigma2)/2;
  
  
   //mexPrintf("\n E: %f \n",*E);

  
  hipMemcpy(Pt1, d_Pt1, M*sizeof(float), hipMemcpyDeviceToHost);  
  
  hipMemcpy(Px, d_Px, M*D*sizeof(float), hipMemcpyDeviceToHost);  
  
  hipMemcpy(P1, d_P1, N* sizeof(float), hipMemcpyDeviceToHost);  

  // Free Device Space, so MATLAB doesnt crash
  hipFree(d_X);
  hipFree(d_Y);
  hipFree(d_PSlice);
  hipFree(d_P1);
  hipFree(d_P1_tmp);
  hipFree(d_Pt1);
  hipFree(d_Px);
  hipFree(d_E);
  hipFree(d_ones);
  hipFree(d_sp);
  
  hipFree(d_denom);
  hipFree(d_X_tmp);

  timer.end();  float t = timer.get();  // elapsed time in seconds
  
  mexPrintf("\n GPU Time: %f ms \n",t * 1000);
  
  //for (n=0; n < N; n++) {
	  //mexPrintf ("\n"); // use for printing P[m]

//       sp=0;
//       for (m=0; m < M; m++) { //iterate through all points (M: width of y)
//           razn=0;
//           for (d=0; d < D; d++) { //iterate through D dimensions
//              diff=x[n+d*N] - y[m+d*M];// *(x+n+d*N)-*(y+m+d*M);  
//              diff=diff*diff; //take the square of the euclidean norm of one scalar -> square the scalar
//              razn+=diff; //proposed name: eucl_dist_sqr; add up the differences for each dimension to get the scalar length of the high-dimensional vector
//           }
// 
//           P[m]=exp(razn/ksig); //nominator
//           
//           sp+=P[m]; //sum in the denominator
//       }
      
      
      //sp+=outlier_tmp; //for this particular x point, we calculate the complete denominator
      
      // Test out if everything works with Pt1 from GPU
      //Pt1[n] = 1-outlier_tmp/ sp; //see documentation: (1 - ca)
      
//       if((float)slice_tmp[n] != (float)(1.0f/sp)){
//     	  
//         	  mexPrintf("Assertion failed! %d - denom[n] on GPU/CPU: %f - %f \n",n, slice_tmp[n],  1.0f/sp);
// //          mexPrintf("sp on CPU: %f \n",sp);
//       }
      
//       for (d=0; d < D; d++) {
//        temp_x[d]=x[n+d*N]/ sp;
//       }
//          
//       for (m=0; m < M; m++) {
//          
//           P1[m]+=P[m]/ sp; //P1 is the P * sum_vector from the equation (see documentation) 
//           
//           for (d=0; d < D; d++) {
//             Px[m+d*M]+= temp_x[d]*P[m]; 
//           }
//           
//       }

      
   //*E +=  -log(sp);	//entropy: measure of overall change
  //}
  
  // Test for P1
//   for (int i = 0; i < M; i++) {
//      for (int d =0; d < D; d++) {
//      if((float)slice_tmp[IDX2C(i,d,M)] != (float)Px[IDX2C(i,d,M)]){
//     	  
//        mexPrintf("Assertion failed! %d - Px[n] on GPU/CPU: %f - %f \n",i,slice_tmp[IDX2C(i,d,M)], Px[IDX2C(i,d,M)]);
// //          mexPrintf("sp on CPU: %f \n",sp);
//         }
//      }
//   }
//   
  //*E +=D*N*log(*sigma2)/2;
  
  free((void*)P);
  free((void*)PSlice);
  free((void*)temp_x);
  free((void*)ones);
  free((void*)filler);
  free((void*)slice_tmp);
  return;
}

/* Input arguments */
#define IN_x		prhs[0]
#define IN_y		prhs[1]
#define IN_sigma2	prhs[2]
#define IN_outlier	prhs[3]


/* Output arguments */
#define OUT_P1		plhs[0]
#define OUT_Pt1		plhs[1]
#define OUT_Px		plhs[2]
#define OUT_E		plhs[3]


/* Gateway routine */
void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[] )
{
    hipError_t cudaStat;    
    hipblasStatus_t stat;
    hipblasHandle_t handle;
	
  float *x, *y, *sigma2, *outlier, *P1, *Pt1, *Px, *E;
  int     N, M, D;
  
  /* Get the sizes of each input argument */
  N = mxGetM(IN_x);
  M = mxGetM(IN_y);
  D = mxGetN(IN_x);
  
  /* Create the new arrays and set the output pointers to them */
  OUT_P1     = mxCreateNumericMatrix(M, 1, mxSINGLE_CLASS, mxREAL);
  OUT_Pt1    = mxCreateNumericMatrix(N, 1, mxSINGLE_CLASS, mxREAL);
  OUT_Px     = mxCreateNumericMatrix(M, D, mxSINGLE_CLASS, mxREAL);
  OUT_E      = mxCreateNumericMatrix(1, 1, mxSINGLE_CLASS, mxREAL);

    /* Assign pointers to the input arguments */
  x      = (float*) mxGetData(IN_x);
  y       = (float*) mxGetData(IN_y);
  sigma2       = (float*) mxGetData(IN_sigma2);
  outlier    = (float*) mxGetData(IN_outlier);
  
  /* Assign pointers to the output arguments */
  P1      = (float*) mxGetData(OUT_P1);
  Pt1      = (float*) mxGetData(OUT_Pt1);
  Px      = (float*) mxGetData(OUT_Px);
  E     = (float*) mxGetData(OUT_E);
   
  /* Do the actual computations in a subroutine */
  cpd_comp(x, y, sigma2, outlier, P1, Pt1, Px, E, N, M, D);
  
  return;
}



