 
/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include ormqr_example.cpp 
 *   nvcc -o a.out ormqr_example.o -L/usr/local/cuda/lib64 -lcublas -lcusolver
 *
 */



#include <stdio.h>
#include "mex.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdlib.h>
#include <assert.h>
#include <hipsolver.h>
using namespace std;

/* Input arguments */
#define IN_A		prhs[0]
#define IN_B		prhs[1]

/* Output arguments */
#define OUT_X		plhs[0]



/* Gateway routine */
void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[] )
{
	double *A, *B, *XC;
	
	/* Get the sizes of each input argument */
	const int N = mxGetM(IN_A);
	const int D = mxGetN(IN_B);
	
	/* Create the new arrays and set the output pointers to them */
	OUT_X     = mxCreateDoubleMatrix(N, D, mxREAL);
	//OUT_X     = mxCreateDoubleMatrix(N, N, mxREAL);
	
	
	/* Assign pointers to the input arguments */
	A      = mxGetPr(IN_A);
	B      = mxGetPr(IN_B);
	
	/* Assign pointers to the output arguments */
	XC      = mxGetPr(OUT_X);
	
	
	//cudsHandle_t cudenseH = NULL;
	hipsolverHandle_t cudenseH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;    
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    
    const int lda = N;
    const int ldb = N;
    const int nrhs1 = D; // number of right hand side vectors
/*       | 1 2 3 |
 *   A = | 4 5 6 |
 *       | 2 1 1 |
 *
 *   x = (1 1 1)'
 *   b = (6 15 4)'
 */
 

    //double A[lda*m] = { 1.0, 4.0, 2.0, 2.0, 5.0, 1.0, 3.0, 6.0, 1.0}; 
//    double X[ldb*nrhs1] = { 1.0, 1.0, 1.0}; // exact solution
    //double B[ldb*nrhs1] = { 6.0, 15.0, 4.0}; 
    //double XC[ldb*nrhs1]; // solution matrix from GPU

    double *d_A = NULL; // linear memory of GPU  
    double *d_tau = NULL; // linear memory of GPU 
    double *d_B  = NULL; 
    int *devInfo = NULL; // info in gpu (device copy)
    double *d_work = NULL;
    int  lwork = 0; 

    int info_gpu = 0;

    const double one = 1;

    /*printf("A = (matlab base-1)\n");
    printMatrix(m, m, A, lda, "A");
    printf("=====\n");
    printf("B = (matlab base-1)\n");
    printMatrix(m, nrhs1, B, ldb, "B");
    printf("=====\n");*/

// step 1: create cudense/cublas handle
    cusolver_status = hipsolverDnCreate(&cudenseH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
    
// step 2: copy A and B to device
    cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(double) * lda * N);
    cudaStat2 = hipMalloc ((void**)&d_tau, sizeof(double) * N);
    cudaStat3 = hipMalloc ((void**)&d_B  , sizeof(double) * ldb * nrhs1);
    cudaStat4 = hipMalloc ((void**)&devInfo, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(double) * lda * N   , hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_B, B, sizeof(double) * ldb * nrhs1, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
 
// step 3: query working space of geqrf and ormqr
    cusolver_status = hipsolverDnDgeqrf_bufferSize(
        cudenseH, 
        N, 
        N, 
        d_A, 
        lda, 
        &lwork);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
 
    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

// step 4: compute QR factorization
    cusolver_status = hipsolverDnDgeqrf(
        cudenseH, 
        N, 
        N, 
        d_A, 
        lda, 
        d_tau, 
        d_work, 
        lwork, 
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    // check if QR is good or not
    cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    printf("after geqrf: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);

// step 5: compute Q^T*B
    cusolver_status= hipsolverDnDormqr(
        cudenseH, 
        HIPBLAS_SIDE_LEFT, 
        HIPBLAS_OP_T,
        N, 
        nrhs1, 
        N, 
        d_A, 
        lda,
        d_tau,
        d_B,
        ldb,
        d_work,
        lwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);
 
    // check if QR is good or not
    cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    printf("after ormqr: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);

// step 6: compute x = R \ Q^T*B

    cublas_status = hipblasDtrsm(
         cublasH,
         HIPBLAS_SIDE_LEFT,
         HIPBLAS_FILL_MODE_UPPER,
         HIPBLAS_OP_N, 
         HIPBLAS_DIAG_NON_UNIT,
         N,
         nrhs1,
         &one,
         d_A,
         lda,
         d_B,
         ldb);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(XC, d_B, sizeof(double)*ldb*nrhs1, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    //printf("X = (matlab base-1)\n");
    //printMatrix(m, nrhs1, XC, ldb, "X");

// free resources
    if (d_A    ) hipFree(d_A);
    if (d_tau  ) hipFree(d_tau);
    if (d_B    ) hipFree(d_B);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);


    if (cublasH ) hipblasDestroy(cublasH);   
    if (cudenseH) hipsolverDnDestroy(cudenseH);   

    hipDeviceReset();

    return;
}