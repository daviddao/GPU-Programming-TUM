/**
 * David Dao, Johannes Rausch, Michal Szymczak
 * TU Munich
 * Sep 2015
 */

#include <stdio.h>
#include "mex.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "hipsolver.h"
#include "time.h"
using namespace std;

// cuda error checking
string prev_file = "";
int prev_line = 0;
#define CUDA_CHECK cuda_check(__FILE__,__LINE__)
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        if (prev_line>0) cout << "Previous CUDA call:" << endl << prev_file << ", line " << prev_line << endl;
        hipDeviceReset();
        //exit(0);
    }
    prev_file = file;
    prev_line = line;
}

/*
 *  solve A*x = b by LU with partial pivoting
 *
 */
/* Input arguments */
#define IN_A		prhs[0]
#define IN_B		prhs[1]

/* Output arguments */
//#define OUT_X		plhs[0]

/* Gateway routine */
void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[] )
{
	double *Acopy, *Bcopy;
	double *A, *B;
    hipsolverHandle_t handle;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    int bufferSize = 0;
    int *info = NULL;
    double *buffer = NULL;
    int *ipiv = NULL; // pivoting sequence
    int h_info = 0;
    int n = 0;
    int d = 0;
    int lda = 0;
    
	/* Get the sizes of each input argument */
	n = mxGetM(IN_A);
	d = mxGetN(IN_B);
	lda = n;
	
	  /* Assign pointers to the input arguments */
	Acopy      = mxGetPr(IN_A);
	Bcopy      = mxGetPr(IN_B);

    cusolver_status = hipsolverDnCreate(&handle);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    
    hipsolverDnDgetrf_bufferSize(handle, n, n, (double*)Acopy, lda, &bufferSize); CUDA_CHECK;

    hipMalloc(&info, sizeof(int)); CUDA_CHECK;
    hipMalloc(&buffer, sizeof(double)*bufferSize); CUDA_CHECK;
    hipMalloc(&A, sizeof(double)*lda*n); CUDA_CHECK;
    hipMalloc(&ipiv, sizeof(int)*n); CUDA_CHECK;
    hipMalloc(&B, sizeof(double)*n*d); CUDA_CHECK;
    

    // prepare a copy of A because getrf will overwrite A with L
    hipMemcpy(A, Acopy, sizeof(double)*lda*n, hipMemcpyHostToDevice); CUDA_CHECK;
    hipMemset(info, 0, sizeof(int)); CUDA_CHECK;

    hipsolverDnDgetrf(handle, n, n, A, lda, buffer, ipiv, info); CUDA_CHECK;
    hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost); CUDA_CHECK;

    if ( 0 != h_info ){
        fprintf(stderr, "Error: LU factorization failed\n");
    }

    hipMemcpy(B, Bcopy, sizeof(double)*n*d, hipMemcpyHostToDevice); CUDA_CHECK;
    hipsolverDnDgetrs(handle, HIPBLAS_OP_N, n, d, A, lda, ipiv, B, n, info); CUDA_CHECK;
    hipDeviceSynchronize(); CUDA_CHECK;
    hipMemcpy(Bcopy, B, sizeof(double)*n*d, hipMemcpyDeviceToHost); CUDA_CHECK;
    
    if (info  ) { hipFree(info  );  CUDA_CHECK;}
    if (buffer) { hipFree(buffer);  CUDA_CHECK;}
    if (A     ) { hipFree(A);  	 CUDA_CHECK;}
    if (B     ) { hipFree(B);  	 CUDA_CHECK;}
    if (ipiv  ) { hipFree(ipiv); 	 CUDA_CHECK;}

    return;
}